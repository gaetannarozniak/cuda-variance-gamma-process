#include "hip/hip_runtime.h"
/**************************************************************
Lokman A. Abbas-Turki code

Those who re-use this code should mention in their code
the name of the author above.
***************************************************************/

#include <stdio.h>
#include <hiprand/hiprand_kernel.h>

__device__ float sigmad[10];
__device__ float thetad[10];
__device__ float kappad[10];
__device__ float strd[4];


// Function that catches the error 
void testCUDA(hipError_t error, const char* file, int line) {

	if (error != hipSuccess) {
		printf("There is an error in file %s at line %d\n", file, line);
		exit(EXIT_FAILURE);
	}
}

// Has to be defined in the compilation in order to get the correct value of the 
// macros __FILE__ and __LINE__
#define testCUDA(error) (testCUDA(error, __FILE__ , __LINE__))

// Set the state for each thread
__global__ void init_curand_state_k(hiprandState* state)
{
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	hiprand_init(0, idx, 0, &state[idx]);
}

// Monte Carlo simulation kernel
// __global__ void MC_k(float dt, float T, int Ntraj, hiprandState* state, float* sum, int* num){

// 	int pidx, same, numR;
// 	float t, X, Y;
// 	int idx = blockDim.x * blockIdx.x + threadIdx.x; 
// 	hiprandState localState = state[idx];
// 	float2 G;
// 	float B;
// 	float price;
// 	float sumR = 0.0f;
// 	float sum2R = 0.0f;
// 	same = idx;

// 	pidx = same/(1296 * 8);
// 	float StrR = Strd[pidx];
// 	same -= (pidx* 1296 * 8);
// 	pidx = same/(216 * 8);
// 	float mR = md[pidx];
// 	same -= (pidx* 216 * 8);
// 	pidx = same/(216);
// 	float alphaR = alphad[pidx];
// 	same -= (pidx*216);
// 	pidx = same/(36);
// 	float betaR = sqrtf(2.0f*alphaR*nu2d[pidx])*(1.0f - expf(mR));//betad[pidx];
// 	same -= (pidx*36);
// 	pidx = same/(6);
// 	float rhoR = rhod[pidx];
// 	same -= (pidx*6);
// 	pidx = same;

// 	numR = 0;
// 	for (int i = 0; i < Ntraj; i++) {
// 		t = 0.0f;
// 		X = 1.0f;
// 		Y = Y0d[pidx];
// 		while(t<T){
// 			G = hiprand_normal2(&localState);
// 			X *= (1.0f + expf(Y)*G.x*dt);
// 			B = rhoR*G.x + sqrtf(1.0f-rhoR*rhoR)*G.y;
// 			Y = Y + alphaR*(mR-Y)*dt*dt + betaR*dt*B;
// 			t += dt*dt;
// 		}
// 		if (X < 12.0f) {
// 			price = fmaxf(0.0f, X - StrR) / Ntraj;
// 			sumR += price;
// 			sum2R += price * price * Ntraj;
// 			numR++;
// 		}
// 	}
// 	sum[2*idx] = sumR*((float)Ntraj/numR);
// 	sum[2*idx + 1] = sum2R*((float)Ntraj / numR);
// 	num[idx] = numR;


// 	/* Copy state back to global memory */
// 	state[idx] = localState;
// }


__device__ float gammaRand(float a, float b, hiprandState *state)
{
    // Handle invalid shape gracefully:
    if (a <= 0.0f) {
        // Could return 0, or assert, etc. 
        // We'll just return 0 here to avoid NaNs.
        return 0.0f;
    }

    // -------------------------------------------------------------------------
    // CASE 1:  0 < a < 1   (Johnk’s algorithm, from homework Algorithm 6.7)
    // -------------------------------------------------------------------------
    if (a < 1.0f)
    {
        // We will produce Gamma(a, 1) and then scale by 1/b to get Gamma(a, b).
        // Johnk’s accept–reject:
        while(true)
        {
            float U = hiprand_uniform(state);
            float V = hiprand_uniform(state);

            // X ~ U^(1/a),  Y ~ V^(1/(1-a))
            float X = powf(U, 1.0f / a);
            float Y = powf(V, 1.0f / (1.0f - a));

            if (X + Y <= 1.0f)
            {
                // Once accepted, generate an exponential(1) deviate E = -ln(U2)
                float E = -__logf(hiprand_uniform(state));
                // This yields a Gamma(a,1) sample = (X/(X+Y)) * E
                float G = (X / (X + Y)) * E;  
                // Finally convert to Gamma(a, b) => scale by 1/b
                return G * (1.0f / b);
            }
        }
    }
    // -------------------------------------------------------------------------
    // CASE 2:  a >= 1   (Best’s algorithm, from homework Algorithm 6.8)
    // -------------------------------------------------------------------------
    else
    {
        // Also known as the “Cheng–Best” or “BC” method
        // We'll generate Gamma(a,1) and then multiply by 1/b => Gamma(a,b).

        float a_minus1 = a - 1.0f;
        float c_       = 3.0f * a - 0.75f;  

        while(true)
        {
            float U = hiprand_uniform(state);
            float V = hiprand_uniform(state);

            // W = U(1-U).  Because 0 < U < 1, W in (0, 0.25].
            float W = U * (1.0f - U);
            // Y ~ +/- sqrt( c_/W ) * (U - 0.5). 
            // This is the “stretch” transform.
            float Y = __fsqrt_rn(c_ / W) * (U - 0.5f);
            // X ~ (a-1) + Y
            float X = a_minus1 + Y;

            // Accept only if X >= 0.
            if (X >= 0.0f)
            {
                // Z = 64 W^3 V^3
                float Z = 64.0f * W * W * W * V * V * V;
                // Now do the acceptance checks:
                // 1) quick reject if Z > (1 - 2Y^2 / X)
                float twoY2overX = 2.0f * Y * Y / X; 
                if (Z <= (1.0f - twoY2overX))
                {
                    // This is a direct accept => X is Gamma(a,1)
                    return X * (1.0f / b);
                }
                // 2) otherwise, do the log check:
                if (__logf(Z) <= 2.0f * (a_minus1 * __logf(X / a_minus1) - Y))
                {
                    // accepted => X is Gamma(a,1)
                    return X * (1.0f / b);
                }
            }
            // else reject and repeat
        }
    }
}
////////////////////////////////////////////////////////////////////////////////
// Kernel: Variance-Gamma simulation for call-price payoff
//   dt   = T / (number_of_steps)
//   Ntraj = # paths per thread
//   wVG  = log((1 - theta*kappa - 0.5*sigma^2*kappa)) / kappa
////////////////////////////////////////////////////////////////////////////////
__global__ void MC_VG(
    float kappa, 
    float theta, 
    float sigma,
    float wVG,      // the drift correction
    float T,        // final maturity
    float K,        // strike
    int   Nsteps,   // how many time steps for the VG discretization
    int   Ntraj,    // how many paths per thread
    hiprandState* state, 
    float* sums     // accumulate payoff sums here
)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    hiprandState localState = state[idx];

    float dt  = T / (float)Nsteps;
    // shape for Gamma is (dt / kappa)
    // scale for Gamma is kappa
    float shape = dt / kappa;
    float scale = kappa;

    float payoffSum  = 0.0f;  // accumulate payoff
    // optional: track sum of squares if you want confidence intervals
    float payoffSum2 = 0.0f;

    // Main loop over # of paths assigned to this thread
    for(int i = 0; i < Ntraj; i++)
    {
        // We will accumulate X_t from 0..T, then exponentiate
        float X = 0.0f;

        // Simulate the partial sums for X(t) from t=0..T
        for(int n = 0; n < Nsteps; n++)
        {
            // 1) Sample Gamma increment:
            float dS = gammaRand(shape, scale, &localState);

            // 2) Sample Normal(0,1):
            float N  = hiprand_normal(&localState);

            // 3) Variance Gamma increment:
            float dX = sigma * N * __fsqrt_rn(kappa * dS)
                       + theta * dS;
            X += dX;
        }

        // final asset price at T:  Y_T = exp( wVG*T + X )
        float Y = __expf(wVG * T + X);

        // payoff of a call option:
        float payoff = fmaxf(Y - K, 0.0f);

        payoffSum  += payoff;
    }

    // store partial sums (later you can reduce across threads)
    sums[idx] = payoffSum;    // \sum of payoffs

    // save state
    state[idx] = localState;
}


int main(void) {
	float sigma[10] = { 0.11f, 0.12f, 0.13f, 0.14f, 0.15f, 0.16f, 0.17f, 0.18f, 0.19f, 0.2f };
	float theta[10] = { -0.34f, -0.3f, -0.27f, -0.24f, -0.21f, -0.18f, -0.15f, -0.12f, -0.09f, -0.06f };
	float kappa[10] = { 0.11f, 0.12f, 0.13f, 0.14f, 0.15f, 0.16f, 0.17f, 0.18f, 0.19f, 0.2f };
	float str[4] = { 100.0f, 95.0f, 90.0f, 85.0f };

	float Tmt[4] = { 3.0f / 12.0f, 6.0f / 12.0f, 1.0f, 2.0f };

	hipMemcpyToSymbol(HIP_SYMBOL(sigmad), sigma, 10 * sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(thetad), theta, 10 * sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(kappad), kappa, 10 * sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(strd), str, 4 * sizeof(float));

	int pidx, same;
	int NTPB = 32;
	int NB =  125;
	int Ntraj = 40000; 
	float dt = 1.0f / (64.0f * 24.0f);
	float strR, kappaR, sigmaR, thetaR, expected_payoff;

	hiprandState* states;
	hipMalloc(&states, NB*NTPB*sizeof(hiprandState));
	init_curand_state_k <<<NB, NTPB>>> (states);
	float *sum;
	hipMallocManaged(&sum, NB*NTPB*sizeof(float));
	FILE* fpt;

	char strg[30];
	for(int i=0; i<4; i++){
		MC_k<<<NB,NTPB>>>(dt, Tmt[i], Ntraj, states, sum);
		hipDeviceSynchronize();
		sprintf(strg, "Tmt%.4f.csv", Tmt[i]);
		fpt = fopen(strg, "w+");
		fprintf(fpt, "sigma, theta, kappa, Str, expected_payoff, Ntraj\n");
		for(int k=0; k< 10 * 10 * 10 * 4; k++){
			expected_payoff = sum[k] / Ntraj;
			strR = str[k % 4];
			kappaR = kappa[(k / 4) % 10];
			thetaR = theta[(k / 40) % 10];
			sigmaR = sigma[(k / 400) % 10];
			fprintf(fpt, "%f, %f, %f, %f, %f, %f, %f, %d\n", sigmaR, thetaR, kappaR, strR, expected_payoff, Ntraj);
		}
		fclose(fpt);
	}
	hipFree(states);
	hipFree(sum);
	hipFree(num);

	return 0;
}